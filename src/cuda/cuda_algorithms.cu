
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for license information.
 */

#ifdef USE_CUDA_EXP

#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

template <typename T>
__global__ void ShufflePrefixSumGlobalKernel(T* values, size_t len, T* block_prefix_sum_buffer) {
  __shared__ T shared_mem_buffer[32];
  const size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  T value = 0;
  if (index < len) {
    value = values[index];
  }
  const T prefix_sum_value = ShufflePrefixSum<T>(value, shared_mem_buffer);
  values[index] = prefix_sum_value;
  if (threadIdx.x == blockDim.x - 1) {
    block_prefix_sum_buffer[blockIdx.x] = prefix_sum_value;
  }
}

template <typename T>
__global__ void ShufflePrefixSumGlobalReduceBlockKernel(T* block_prefix_sum_buffer, int num_blocks) {
  __shared__ T shared_mem_buffer[32];
  const int num_blocks_per_thread = (num_blocks + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 2) / (GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1);
  int thread_block_start = threadIdx.x == 0 ? 0 : (threadIdx.x - 1) * num_blocks_per_thread;
  int thread_block_end = threadIdx.x == 0 ? 0 : min(thread_block_start + num_blocks_per_thread, num_blocks);
  T base = 0;
  for (int block_index = thread_block_start; block_index < thread_block_end; ++block_index) {
    base += block_prefix_sum_buffer[block_index];
  }
  base = ShufflePrefixSum<T>(base, shared_mem_buffer);
  thread_block_start = threadIdx.x == blockDim.x - 1 ? 0 : threadIdx.x * num_blocks_per_thread;
  thread_block_end = threadIdx.x == blockDim.x - 1 ? 0 : min(thread_block_start + num_blocks_per_thread, num_blocks);
  for (int block_index = thread_block_start + 1; block_index < thread_block_end; ++block_index) {
    block_prefix_sum_buffer[block_index] += block_prefix_sum_buffer[block_index - 1];
  }
  for (int block_index = thread_block_start; block_index < thread_block_end; ++block_index) {
    block_prefix_sum_buffer[block_index] += base;
  }
}

template <typename T>
__global__ void ShufflePrefixSumGlobalAddBase(size_t len, const T* block_prefix_sum_buffer, T* values) {
  const T base = blockIdx.x == 0 ? 0 : block_prefix_sum_buffer[blockIdx.x - 1];
  const size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (index < len) {
    values[index] += base;
  }
}

template <typename T>
void ShufflePrefixSumGlobalInner(T* values, size_t len, T* block_prefix_sum_buffer) {
  const int num_blocks = (static_cast<int>(len) + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1) / GLOBAL_PREFIX_SUM_BLOCK_SIZE;
  ShufflePrefixSumGlobalKernel<<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(values, len, block_prefix_sum_buffer);
  ShufflePrefixSumGlobalReduceBlockKernel<<<1, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(block_prefix_sum_buffer, num_blocks);
  ShufflePrefixSumGlobalAddBase<<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(len, block_prefix_sum_buffer, values);
}

template <>
void ShufflePrefixSumGlobal(uint16_t* values, size_t len, uint16_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint16_t>(values, len, block_prefix_sum_buffer);
}

template <>
void ShufflePrefixSumGlobal(uint32_t* values, size_t len, uint32_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint32_t>(values, len, block_prefix_sum_buffer);
}

template <>
void ShufflePrefixSumGlobal(uint64_t* values, size_t len, uint64_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint64_t>(values, len, block_prefix_sum_buffer);
}

template <typename T>
__global__ void BlockReduceSum(T* block_buffer, const data_size_t num_blocks) {
  __shared__ T shared_buffer[32];
  T thread_sum = 0;
  for (data_size_t block_index = static_cast<data_size_t>(threadIdx.x); block_index < num_blocks; block_index += static_cast<data_size_t>(blockDim.x)) {
    thread_sum += block_buffer[block_index];
  }
  thread_sum = ShuffleReduceSum<T>(thread_sum, shared_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    block_buffer[0] = thread_sum;
  }
}

template <typename VAL_T, typename REDUCE_T>
__global__ void ShuffleReduceSumGlobalKernel(const VAL_T* values, const data_size_t num_value, REDUCE_T* block_buffer) {
  __shared__ REDUCE_T shared_buffer[32];
  const data_size_t data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  const REDUCE_T value = (data_index < num_value ? static_cast<REDUCE_T>(values[data_index]) : 0.0f);
  const REDUCE_T reduce_value = ShuffleReduceSum<REDUCE_T>(value, shared_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    block_buffer[blockIdx.x] = reduce_value;
  }
}

template <typename VAL_T, typename REDUCE_T>
void ShuffleReduceSumGlobalInner(const VAL_T* values, size_t n, REDUCE_T* block_buffer) {
  const data_size_t num_value = static_cast<data_size_t>(n);
  const data_size_t num_blocks = (num_value + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1) / GLOBAL_PREFIX_SUM_BLOCK_SIZE;
  ShuffleReduceSumGlobalKernel<VAL_T, REDUCE_T><<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(values, num_value, block_buffer);
  BlockReduceSum<REDUCE_T><<<1, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(block_buffer, num_blocks);
}

template <>
void ShuffleReduceSumGlobal<label_t, double>(const label_t* values, size_t n, double* block_buffer) {
  ShuffleReduceSumGlobalInner(values, n, block_buffer);
}

template <typename VAL_T, typename REDUCE_T>
__global__ void ShuffleReduceDotProdGlobalKernel(const VAL_T* values1, const VAL_T* values2, const data_size_t num_value, REDUCE_T* block_buffer) {
  __shared__ REDUCE_T shared_buffer[32];
  const data_size_t data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  const REDUCE_T value1 = (data_index < num_value ? static_cast<REDUCE_T>(values1[data_index]) : 0.0f);
  const REDUCE_T value2 = (data_index < num_value ? static_cast<REDUCE_T>(values2[data_index]) : 0.0f);
  const REDUCE_T reduce_value = ShuffleReduceSum<REDUCE_T>(value1 * value2, shared_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    block_buffer[blockIdx.x] = reduce_value;
  }
}

template <typename VAL_T, typename REDUCE_T>
void ShuffleReduceDotProdGlobalInner(const VAL_T* values1, const VAL_T* values2, size_t n, REDUCE_T* block_buffer) {
  const data_size_t num_value = static_cast<data_size_t>(n);
  const data_size_t num_blocks = (num_value + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1) / GLOBAL_PREFIX_SUM_BLOCK_SIZE;
  ShuffleReduceDotProdGlobalKernel<VAL_T, REDUCE_T><<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(values1, values2, num_value, block_buffer);
  BlockReduceSum<REDUCE_T><<<1, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(block_buffer, num_blocks);
}

template <>
void ShuffleReduceDotProdGlobal<label_t, double>(const label_t* values1, const label_t* values2, size_t n, double* block_buffer) {
  ShuffleReduceDotProdGlobalInner(values1, values2, n, block_buffer);
}

}  // namespace LightGBM

#endif  // USE_CUDA_EXP